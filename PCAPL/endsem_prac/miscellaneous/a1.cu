
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void matrixKernel(int *matrix, int *result){
    int row = threadIdx.x;
    int col = threadIdx.y;
    int m = blockDim.x;
    int n = blockDim.y;
    int sum = 0;
    for(int i = 0; i < n; i++) sum += matrix[row*n+i];
    for(int i = 0; i < m; i++) sum += matrix[i*n+col];
    result[row*n+col] = sum - matrix[row*n+col];
}

int main(){

    int m, n;
    printf("Enter m and n : ");
    scanf("%d %d", &m, &n);
    int size = m*n*sizeof(int);
    int *matrix = (int*)malloc(size);
    int *result = (int*)malloc(size);
    printf("Enter elements of matrix : \n");
    for(int i = 0; i < m*n; i++) scanf("%d", &matrix[i]);

    int *d_matrix, *d_result;
    hipMalloc((void**)&d_matrix, size);
    hipMalloc((void**)&d_result, size);
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    dim3 dimBlock(m,n,1);
    matrixKernel<<<1, dimBlock>>>(d_matrix, d_result);

    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    printf("Result : ");
    for(int i = 0; i < m; i++){
        printf("\n");
        for(int j = 0; j < n; j++)
            printf("%d ", result[i*n+j]);
    }

    hipFree(d_matrix);
    hipFree(d_result);

    return 0;

}