#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#include"hip/hip_runtime.h"

__global__ void change(char *a, char *b, int size1){
    int tid=threadIdx.x;
    int start = tid * size1 - (tid * (tid -1))/2;
    for(int i=0;i<size1-tid;i++){
        b[start]=a[i];
        start++;
    }
}

int main(void){
    char a[100],b[100];
    printf("Enter the sentence\n");
    scanf("%[^\n]s",a);
    int n1=strlen(a);
    int size1=n1 *sizeof(char);
    int n2 = n1 * (n1 + 1) /2;
    int size2=n2* sizeof(char);

    char *d_a, *d_b;
    hipMalloc((void**)&d_a, size1);
    hipMalloc((void**)&d_b,size2);

    hipMemcpy(d_a,a, size1,hipMemcpyHostToDevice);
    change<<<1,n1>>>(d_a,d_b,n1);
    hipMemcpy(b,d_b, size2,hipMemcpyDeviceToHost);
    printf("The result is %s\n",b);

    return 0; 

}