#include<stdio.h>
#include<string.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void change(char *a,char *b,int size)
{
	int tid=threadIdx.x;
    int start = 0;
    for (int i = 0; i < tid; i++)
        start += size-i;
    for(int i=0;i<size-tid;i++)
        b[start++]=a[i];
}
int main(void)
{
	char a[100],b[100];
	char *d_a,*d_b;
	printf("Enter string: ");
	scanf("%[^\n]%*c",a);
	int n1=strlen(a);
	int size1=n1*sizeof(char);
    int n2=n1;
    for(int i=n1;i>0;i--)
        n2+=i;    
    int size2=n2*sizeof(char);    
	hipMalloc((void **)&d_a,size1);
	hipMalloc((void **)&d_b,size2);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	change<<<1,n1>>>(d_a,d_b,size1);
	hipMemcpy(b,d_b,size2,hipMemcpyDeviceToHost);
	printf("Result: %s\n",b);	
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}