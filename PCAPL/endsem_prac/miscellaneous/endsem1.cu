#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h> 
#include "hip/hip_runtime.h"
#include ""

// <<< 1, length >>>
__global__ void rightRotation(char *string, char *result, int length, int n){
    int tid = threadIdx.x;
    if(tid < length - n)
        result[tid + n] = string[tid];
    else
        result[length - tid - 1] = string[tid];
}

// // <<< 1, length >>>
// __global__ void leftRotation(char *string, char *result, int length, int n){
//     int tid = threadIdx.x;
//     if(tid < n)
//         result[length - n + tid] = string[tid];
//     else
//         result[tid - n] = string[tid];
// }

// <<< 1, (2,2,1) >>>
__global__ void rightRotation2D(char *string, char *result, int length, int n){
    int tid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    if(tid < length){
        if(string[tid] == ' ') result[tid] = ' ';
        else if((tid == 0) || (tid > 0 && string[tid - 1] == ' ')){
            int end = tid; 
            while(string[end + 1] != ' ' && end < len - 1)
                end++; 
            int wordlen = end - tid + 1; 
            for(int i = tid; i <= end; i++)
                result[]
        }
    }
}

int main(){
    char string[100], temp[100], *d_string, *result, *d_result; 
    int length, n; 

    printf("Enter the string: ");
    scanf("%[^\n]s",string);

    length = strlen(string);

    printf("Enter value of n: ");
    scanf("%d", &n);

    result = (char *)malloc(length*sizeof(char));

    hipMalloc((void **)&d_string, length*sizeof(char));
    hipMalloc((void **)&d_result, length*sizeof(char));

    printf("Result: \n");

    for(int i = 0; i < length; i++){
        int j = 0;
        while(string[i] != ' ')
            temp[j++] = string[i++];

        hipMemcpy(d_string, temp, j*sizeof(char), hipMemcpyHostToDevice);

        rightRotation <<< 1, j >>> (d_string, d_result, j, n);

        hipMemcpy(result, d_result, length*sizeof(char), hipMemcpyDeviceToHost);

        printf("%s\n", result);
    }

    // dim3 dimBlock(2, 2, 1);

    // rightRotation2D <<< 1, dimBlock >>> (d_string, d_result, length, n);

    // hipMemcpy(result, d_result, length*sizeof(char), hipMemcpyDeviceToHost);

    // printf("Rotated string: %s\n", result);

    // leftRotation <<< 1, length >>> (d_string, d_result, length, n);

    // hipMemcpy(result, d_result, length*sizeof(char), hipMemcpyDeviceToHost);

    // printf("Rotated string: %s\n", result);

    // hipFree(d_string);
    // hipFree(d_result);

    return 0; 
}