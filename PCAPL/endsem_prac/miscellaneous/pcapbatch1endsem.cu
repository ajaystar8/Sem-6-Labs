#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void leftRotate(char *d_word, char *d_ans, int index, int len) {
    int tid = (blockIdx.x*blockDim.x*blockDim.y) + (blockDim.x*threadIdx.y) + threadIdx.x;
    if((tid<len) && d_word[tid] == ' ') d_ans[tid] = ' ';
    else if((tid<len) && tid == 0 || (tid>0 && d_word[tid-1] == ' ')) {
        int end = tid;
        while((end < (len-1)) && (d_word[end+1] != ' ')) end++;
        int lenofword = end - tid + 1;
        for(int i=tid; i<=end; i++)
            d_ans[((i-tid-index+lenofword)%lenofword)+tid] = d_word[i];
    }
}

__global__ void rightRotate(char *d_word, char *d_ans, int index, int len) {
    int tid = (blockIdx.x*blockDim.x*blockDim.y) + (blockDim.x*threadIdx.y) + threadIdx.x;
    if((tid<len) && d_word[tid] == ' ') d_ans[tid] = ' ';
    else if((tid<len) && tid == 0 || (tid>0 && d_word[tid-1] == ' ')) {
        int end = tid;
        while((end < (len-1)) && (d_word[end+1] != ' ')) end++;
        int lenofword = end - tid + 1;
        for(int i=tid; i<=end; i++)
            d_ans[((i-tid+index)%lenofword)+tid] = d_word[i];
    }
}

int main() {
    char h_word[100], *h_ans;
    char *d_word, *d_ans;

    int index;
    int now = 0, maxlen = 0;

    printf("Enter the word: ");
    scanf("%[^\n]s", h_word);
    printf("Enter the index: ");
    scanf("%d", &index);

    int len = strlen(h_word);

    int templen = 0;
    for(int i=0; i<len; i++) {
        if(i==0 || h_word[i] == ' ') now++;
        if(h_word[i] != ' ') templen++;
        else {
            if(maxlen < templen) maxlen = templen;
            templen = 0;
        }
    }
    
    h_ans = (char*)malloc(len*sizeof(char));

    hipMalloc((void**)&d_word, len*sizeof(char));
    hipMalloc((void**)&d_ans, len*sizeof(char));

    dim3 dimGrid(now, 1, 1);
    dim3 dimBlock(4, 2, 1);

    hipMemcpy(d_word, h_word, len*sizeof(char), hipMemcpyHostToDevice);

    rightRotate<<<dimGrid, dimBlock>>>(d_word, d_ans, index, len);

    hipMemcpy(h_ans, d_ans, len*sizeof(char), hipMemcpyDeviceToHost);

    printf("The final string after right rotation is: %s\n", h_ans);

    leftRotate<<<1, len>>>(d_word, d_ans, index, len);

    hipMemcpy(h_ans, d_ans, len*sizeof(char), hipMemcpyDeviceToHost);

    printf("The final string after left rotation is: %s\n", h_ans);
    
    return 0;
}