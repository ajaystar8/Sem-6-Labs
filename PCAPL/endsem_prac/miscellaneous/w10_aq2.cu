#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void kernelMatrixOperation(char *d_letters, int *d_numbers, char *d_output) {
    int tid = threadIdx.x;
    int currIdx = 0;

    for(int i=0; i<tid; i++)
        currIdx += d_numbers[i];
    
    for(int j=currIdx; j<currIdx+d_numbers[tid]; j++)
        d_output[j] = d_letters[tid];
}

int main(){
    char *d_letters, *output, *d_output; 
    int *d_numbers, m, n;
    char letters[4] = {'a', 'b', 'c', 'd'};
    int numbers[4] = {1, 2, 3, 4};

    printf("Enter m: ");
    scanf("%d", &m); 

    printf("Enter n: ");
    scanf("%d", &n);

    // letters = (char *)malloc(m*n*sizeof(char));
    // numbers = (int *)malloc(m*n*sizeof(int));
    
    // printf("Enter char matrix: ");
    // for(int i = 0; i < m*n; i++)
    //     scanf("%c", &letters[i]);

    // fflush(stdin);
    
    // printf("Enter int matrix: ");
    // for(int i = 0; i < m*n; i++)
    //     scanf("%d", &numbers[i]);

    int res_size = 1;
    for(int i = 0; i < m*n; i++)
        res_size += numbers[i];

    output = (char *)malloc(res_size*sizeof(char));

    hipMalloc((void **)&d_letters, m*n*sizeof(char));
    hipMalloc((void **)&d_numbers, m*n*sizeof(int));
    hipMalloc((void **)&d_output, res_size*sizeof(char));

    hipMemcpy(d_letters, letters, m*n*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_numbers, numbers, m*n*sizeof(int), hipMemcpyHostToDevice);

    kernelMatrixOperation <<<1, m*n>>> (d_letters, d_numbers, d_output);

    hipMemcpy(output, d_output, res_size*sizeof(char), hipMemcpyDeviceToHost);

    printf("Resultant string: \n");
    for(int i = 0; i < res_size; i++)
        printf("%c", output[i]);

    hipFree(d_letters);
    hipFree(d_numbers);
    hipFree(d_output);
    
    
    return 0; 
}