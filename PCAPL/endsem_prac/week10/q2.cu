#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


// <<<rows, cols>>>
__global__ void rowOperation(int *d_a, int *d_b){
    int rowId = blockIdx.x; 
    int width = blockDim.x;
    int newVal = 1;
    int k = threadIdx.x;
    int power = rowId + 1; 
    while(power){
        newVal *= d_a[rowId * width + k];
        power--;
    }
    d_b[rowId * width + k] = newVal;
}

int main(){
    int *a, *b, m, n, size; 
    int *d_a, *d_b;

    printf("Enter m: ");
    scanf("%d", &m);

    printf("Enter n: ");
    scanf("%d", &n);

    size = m * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);

    printf("Enter the values of the matrix: ");
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    rowOperation <<<m, n>>> (d_a, d_b);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++)
            printf("%d\t", b[i*m + j]); 
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0; 
}