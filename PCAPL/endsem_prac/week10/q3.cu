#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void doOperation(int *d_a, int *d_b, int m, int n){
    int rowId = blockIdx.x;
    int width = blockDim.x; 
    int k = threadIdx.x;

    if((rowId > 0 && rowId < m - 1) && (k > 0 && k < n -1))
        d_b[rowId * width + k] = 2 * d_a[rowId * width + k];
    else
        d_b[rowId * width + k] = d_a[rowId * width + k];
}

int main(){
    int *a, *b, m, n, size; 
    int *d_a, *d_b;

    printf("Enter m: ");
    scanf("%d", &m);

    printf("Enter n: ");
    scanf("%d", &n);

    size = m * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);

    printf("Enter the values of the matrix: ");
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    doOperation <<<m, n>>> (d_a, d_b, m, n);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++)
            printf("%d\t", b[i*m + j]); 
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0; 
}