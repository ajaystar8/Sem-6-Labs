#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


// n,m,1
__global__ void matrixOperation(int *d_a, int *d_b, int m, int n){
    int rid = threadIdx.y;
    int cid = threadIdx.x;
    int sum = 0;
    if(rid < m && cid < n){
        for(int k = 0; k < n; k++)
            sum += d_a[rid * n + k];
        for(int k = 0; k < m; k++)
            sum += d_a[k * n + cid];
        sum -= d_a[rid * n + cid];
    }
    d_b[rid * n + cid] = sum;
}

int main(){
    int *a, *d_a, *b, *d_b, m, n;
    
    printf("Enter m: ");
    scanf("%d", &m); 

    printf("Enter n: ");
    scanf("%d", &n);

    int size = m*n*sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);

    printf("Enter the input matrix: ");
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);
    
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(n,m,1);
    matrixOperation <<<1, dimBlock>>> (d_a, d_b, m, n);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix: \n");

    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d\t", b[i*n + j]); 
        printf("\n");
    }

    return 0; 
}

