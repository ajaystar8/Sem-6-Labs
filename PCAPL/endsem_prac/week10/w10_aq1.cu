#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


// <<< (1,1,1), (n,m,1) >>>
__global__ void kernelOperation(int *d_a, int *d_b, int m, int n){
    int row = threadIdx.y; 
    int col = threadIdx.x; 
    int sum = 0;
    // col sum
    for(int i = 0; i < m; i++)
        sum += d_a[i * n + col];
    // row sum
    for(int i = 0; i < n; i++)
        sum += d_a[row * n + i];
    d_b[row*n + col] = sum;
}

int main(){
    int *a, *b, *d_a, *d_b, m, n, size; 

    printf("Enter m & n: ");
    scanf("%d %d", &m, &n);

    size = m*n*sizeof(int);

    a = (int *)malloc(m*n*sizeof(int));
    b = (int *)malloc(m*n*sizeof(int));

    printf("Enter elements of matrix: ");
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimGrid(1,1,1);
    dim3 dimBlock(n,m,1);
    kernelOperation <<<dimGrid, dimBlock>>> (d_a, d_b, m, n);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix: \n");
    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d ", b[i*n + j]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0; 
}

/*
1 2 3
4 5 6
*/
