#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void matrixOperation(char *d_letters, int *d_numbers, char *d_output){
    int tid = threadIdx.x;

    int startIdx = 0; 
    for(int i = 0; i < tid; i++)
        startIdx += d_numbers[i];
    
    for(int i = startIdx; i < startIdx + d_numbers[tid]; i++)
        d_output[i] = d_letters[tid];
}

int main(){
    char *d_letters, *output, *d_output; 
    int *d_numbers, m, n;
    char letters[4] = {'a', 'b', 'c', 'd'};
    int numbers[4] = {1, 2, 3, 4};

    printf("Enter m: ");
    scanf("%d", &m); 

    printf("Enter n: ");
    scanf("%d", &n);

    int res_size = 0;
    for(int i = 0; i < m*n; i++)
        res_size += numbers[i];

    output = (char *)malloc(res_size*sizeof(char));

    hipMalloc((void **)&d_letters, m*n*sizeof(char));
    hipMalloc((void **)&d_numbers, m*n*sizeof(int));
    hipMalloc((void **)&d_output, res_size*sizeof(char));

    hipMemcpy(d_letters, letters, m*n*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_numbers, numbers, m*n*sizeof(int), hipMemcpyHostToDevice);

    matrixOperation <<<1, m*n>>> (d_letters, d_numbers, d_output);

    hipMemcpy(output, d_output, res_size*sizeof(char), hipMemcpyDeviceToHost);

    printf("Resultant string: \n");
    for(int i = 0; i < res_size; i++)
        printf("%c", output[i]);

    hipFree(d_letters);
    hipFree(d_numbers);
    hipFree(d_output);

    return 0; 
}