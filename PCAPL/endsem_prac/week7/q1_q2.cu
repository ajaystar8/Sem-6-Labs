#include <stdio.h> 
#include <stdlib.h> 
#include "hip/hip_runtime.h"


// // <<< N, 1 >>>
// __global__ void vectorSum1(int *a, int *b, int *c){
//     int elemIdx = blockIdx.x; 
//     c[elemIdx] = a[elemIdx] + b[elemIdx];
// }

// // <<< 1, N >>>
// __global__ void vectorSum2(int *a, int *b, int *c){
//     int elemIdx = threadIdx.x; 
//     c[elemIdx] = a[elemIdx] + b[elemIdx];
// }

// <<< ceil(N/256.0,1,1), (256,1,1) >>>
__global__ void vectorSum3(int *a, int *b, int *c, int n){
    int elemIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(elemIdx < n) 
        c[elemIdx] = a[elemIdx] + b[elemIdx];
}

int main(){
    int *a, *b, *c, *d_a, *d_b, *d_c, n, size;

    printf("Enter number of elements: ");
    scanf("%d", &n);

    size = n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    printf("Enter elements of A: ");
    for(int i = 0; i < n; i++)
        scanf("%d", &a[i]);

    printf("Enter elements of B: ");
    for(int i = 0; i < n; i++)
        scanf("%d", &b[i]);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // vectorSum1 <<< n, 1 >>> (d_a, d_b, d_c);

    // cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

    // printf("Result: \n");
    // for(int i = 0; i < n; i++)
    //     printf("%d ", c[i]);

    // vectorSum2 <<< 1, n >>> (d_a, d_b, d_c);

    // cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

    // printf("Result: \n");
    // for(int i = 0; i < n; i++)
    //     printf("%d ", c[i]);

    dim3 dimGrid(ceil(n/256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);
    vectorSum3 <<< dimGrid, dimBlock >>> (d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result: \n");
    for(int i = 0; i < n; i++)
        printf("%d ", c[i]);

    return 0; 
}