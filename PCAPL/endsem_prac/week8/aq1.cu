#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"


__global__ void stringReverse(char *d_string, int len){
    int start = threadIdx.x; 
    if(start == 0 || d_string[start - 1] == ' '){
        int end = start; 
        while(d_string[end + 1] != ' ' && end + 1 != len)
            end++;
        for(int i = start; i <= (end + start)/2; i++){
            char temp = d_string[end + start - i];
            d_string[end + start - i] = d_string[i];
            d_string[i] = temp; 
        }
    }
}

int main(){
    char string[100], *d_string, *result; 

    printf("Enter string: ");
    scanf("%[^\n]s",string);

    int len = strlen(string);

    result = (char *)malloc(len * sizeof(char));

    hipMalloc((void **)&d_string, len*sizeof(char));

    hipMemcpy(d_string, string, len*sizeof(char), hipMemcpyHostToDevice);

    stringReverse <<< 1, len >>> (d_string, len);

    hipMemcpy(result, d_string, len*sizeof(char), hipMemcpyDeviceToHost);

    printf("Reversed string: %s\n", result);

    hipFree(d_string);

    return 0; 
}