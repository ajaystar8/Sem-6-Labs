#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"


__global__ void stringRepeat(char *d_string, char *d_result, int length, int n){
    int tid = threadIdx.x;
    for(int i = 0; i < n; i++){
        int targetIdx = i * length + tid;
        d_result[targetIdx] = d_string[tid];
    }
}

int main(){
    char string[100], *d_string, *result, *d_result; 
    int n; 

    printf("Enter the string: ");
    scanf("%s", string);

    printf("Enter n: ");
    scanf("%d", &n);

    int length = strlen(string);

    int size = length * sizeof(char);
    int res_size = length * n * sizeof(char);

    result = (char *)malloc(res_size);

    hipMalloc((void **)&d_string, size);
    hipMalloc((void **)&d_result, res_size);

    hipMemcpy(d_string, string, size, hipMemcpyHostToDevice);

    stringRepeat <<< 1, length >>> (d_string, d_result, length, n);

    hipMemcpy(result, d_result, res_size, hipMemcpyDeviceToHost);

    printf("New string: %s\n", result);

    hipFree(d_result);
    hipFree(d_string);

    return 0; 
}