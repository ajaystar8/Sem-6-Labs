#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void stringPatternGenerate(char *d_string, char *d_result){
    int tid = threadIdx.x; 
    int startIdx = tid * (tid + 1) / 2;
    for(int i = 0; i < tid + 1; i++)
        d_result[startIdx + i] = d_string[tid];
}

int main(){
    char string[100], *d_string, *result, *d_result; 

    printf("Enter the string: ");
    scanf("%s", string);

    int length = strlen(string);

    int size = length * sizeof(char);
    int res_size = (length * (length + 1) / 2) * sizeof(char);

    result = (char *)malloc(res_size);

    hipMalloc((void **)&d_string, size);
    hipMalloc((void **)&d_result, res_size);

    hipMemcpy(d_string, string, size, hipMemcpyHostToDevice);

    stringPatternGenerate <<< 1, length >>> (d_string, d_result);

    hipMemcpy(result, d_result, res_size, hipMemcpyDeviceToHost);

    printf("New string: %s\n", result);

    hipFree(d_result);
    hipFree(d_string);

    return 0; 
}