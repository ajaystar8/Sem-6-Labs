#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define MAX_SENTENCE_LENGTH 100
#define MAX_WORD_LENGTH 10
#define BLOCK_SIZE 128

__global__ void wordMatch(char *d_sentence, char *d_word, int *d_wordCount, int len){
    int tid = threadIdx.x; 
    int i = 0; 
    while(i < blockDim.x && d_word[i] != '\0' && d_sentence[tid + i] == d_word[i])
        i++; 
    if(i == len)
        atomicAdd(d_wordCount, 1);
}

int main() {
    char sentence_h[100], word_h[100];
    int wordCount_h = 0;

    printf("Enter a sentence: ");
    scanf("%[^\n]s",sentence_h);
    printf("Enter a word: ");
    scanf("%s", word_h);

    char* sentence_d;
    char* word_d;
    int* wordCount_d;

    hipMalloc(&sentence_d, MAX_SENTENCE_LENGTH * sizeof(char));
    hipMalloc(&word_d, MAX_WORD_LENGTH * sizeof(char));
    hipMalloc(&wordCount_d, sizeof(int));
    
    hipMemcpy(sentence_d, sentence_h, MAX_SENTENCE_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(word_d, word_h, MAX_WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(wordCount_d, &wordCount_h, sizeof(int), hipMemcpyHostToDevice);
    
    wordMatch<<< 1 ,strlen(sentence_h)>>>(sentence_d, word_d, wordCount_d, strlen(word_h));

    hipMemcpy(&wordCount_h, wordCount_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word_h, wordCount_h);

    hipFree(sentence_d);
    hipFree(word_d);
    hipFree(wordCount_d);

    return 0;

}