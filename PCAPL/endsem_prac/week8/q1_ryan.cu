#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX_SENTENCE_LENGTH 100
#define MAX_WORD_LENGTH 10
#define BLOCK_SIZE 128

__global__ void countWordKernel(char* sentence_h, int* wordCount_h, char* word_h, int len) {
    int index = threadIdx.x;
    int i = 0;
    while (i < blockDim.x && word_h[i] != '\0' && sentence_h[index + i] == word_h[i]) {
        i++;
    }
    if (i == len || word_h[i] == '\0') {
        atomicAdd(wordCount_h, 1);
    }
}

int main() {
    char sentence_h[100], word_h[100];
    int wordCount_h = 0;

    printf("Enter a sentence: ");
    scanf("%[^\n]s",sentence_h);
    printf("Enter a word: ");
    scanf("%s", word_h);

    char* sentence_d;
    char* word_d;
    int* wordCount_d;

    hipMalloc(&sentence_d, MAX_SENTENCE_LENGTH * sizeof(char));
    hipMalloc(&word_d, MAX_WORD_LENGTH * sizeof(char));
    hipMalloc(&wordCount_d, sizeof(int));
    
    hipMemcpy(sentence_d, sentence_h, MAX_SENTENCE_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(word_d, word_h, MAX_WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(wordCount_d, &wordCount_h, sizeof(int), hipMemcpyHostToDevice);
    
    countWordKernel<<< 1 ,strlen(sentence_h)>>>(sentence_d, wordCount_d, word_d, strlen(word_h));

    hipMemcpy(&wordCount_h, wordCount_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word_h, wordCount_h);

    hipFree(sentence_d);
    hipFree(word_d);
    hipFree(wordCount_d);

    return 0;

}
