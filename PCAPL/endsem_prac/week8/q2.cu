#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


/*
len = 4
tid = 0 => startidx = 0
tid = 1 => startidx = 0 + 4 = 4 = 4 - 0
tid = 2 => startidx = 0 + 4 + 3 = 7 = 4*2 - 1
tid = 3 => startidx = 0 + 4 + 3 + 2 = 9 = 4*3 - 3
*/

//<<<1, len(string)>>>
__global__ void stringKernel(char *word, char *result){
    int tid = threadIdx.x; 
    int len = blockDim.x; 
    int start = (tid * len) - (tid * (tid - 1))/2;
    for(int i = 0; i < len - tid; i++)
        result[i + start] = word[i];
}

int main(){
    char *word, *result, *d_word, *d_result;
    int len; 

    printf("Enter word length: ");
    scanf("%d", &len);

    word = (char *)malloc(len*sizeof(char));
    result = (char *)malloc((len * (len+1)/2)*sizeof(char));

    printf("Enter string: ");
    scanf("%s", word);

    hipMalloc((void **)&d_word, len * sizeof(char));
    hipMalloc((void **)&d_result, (len * (len+1)/2)*sizeof(char));

    hipMemcpy(d_word, word, len*sizeof(char), hipMemcpyHostToDevice);

    stringKernel <<<1, len>>> (d_word, d_result);

    hipMemcpy(result, d_result, (len * (len+1)/2)*sizeof(char), hipMemcpyDeviceToHost);

    printf("Resultant string: ");
    for(int i = 0; i < (len * (len+1)/2); i++)
        printf("%c ", result[i]);

    hipFree(d_word);
    hipFree(d_result);

    return 0; 
}