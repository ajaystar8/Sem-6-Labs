
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void copyKernel(char *word, char *result){
    int len = blockDim.x;
    int tid = threadIdx.x;
    int start = (tid*len)-(((tid-1)*tid)/2);
    for(int i = 0; i < len-tid; i++) result[start+i] = word[i];
}

int main(){

    char word[100];
    printf("Enter word : ");
    scanf("%s", word);
    int len = strlen(word);
    int reslen = (len*(len+1))/2;
    char *result = (char*)malloc((reslen+1)*sizeof(char));

    char *d_word, *d_result;
    hipMalloc((void**)&d_word, (len+1)*sizeof(char));
    hipMalloc((void**)&d_result, (reslen+1)*sizeof(char));

    hipMemcpy(d_word, word, len*sizeof(char), hipMemcpyHostToDevice);
    copyKernel<<<1, len>>>(d_word, d_result);

    hipMemcpy(result, d_result, reslen*sizeof(char), hipMemcpyDeviceToHost);

    printf("Result : %s\n", result);

    return 0;
}