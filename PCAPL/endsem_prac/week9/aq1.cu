#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

// n,m
__global__ void kernelOperation(int *d_a, int *d_b, int m, int n){
    int cid = threadIdx.x;
    int rid = threadIdx.y;
    if(rid < m && cid < n){
        int elem = d_a[rid*n + cid];
        if(elem % 2){
            // odd number
            // col sum
            int sum = 0; 
            for(int k = 0; k < m; k++) 
                sum += d_a[k * n + cid];
            d_b[rid*n + cid] = sum;
        }
        else{
            // even number
            // row sum
            int sum = 0; 
            for(int k = 0; k < n; k++) 
                sum += d_a[rid * n + k];
            d_b[rid * n + cid] = sum;
        }
    }
}

int main(){
    int *a, *d_a, *b, *d_b, m, n;
    
    printf("Enter m: ");
    scanf("%d", &m); 

    printf("Enter n: ");
    scanf("%d", &n);

    int size = m*n*sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);

    printf("Enter the input matrix: ");
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);
    
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(n,m,1);
    kernelOperation <<<1, dimBlock>>> (d_a, d_b, m, n);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix: \n");

    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++)
            printf("%d\t", b[i*n + j]); 
        printf("\n");
    }

    return 0; 
}