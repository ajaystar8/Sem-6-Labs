#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

// 1, n,m
__global__ void matrixKernel(int *d_a, int *d_b, int n){
    int rid = threadIdx.y; 
    int cid = threadIdx.x; 
    if (rid < n && cid < n){
        if(rid == cid)
            d_b[rid * n + cid] = 0;
        else if(rid > cid){
            // lower traingle
            int sum = 0; 
            int elem = d_a[rid * n + cid];
            while(elem){
                sum += elem % 10; 
                elem /= 10;
            }
            d_b[rid * n + cid] = sum; 
        }
        else{
            // upper traingle
            int fact = 1; 
            int elem = d_a[rid * n + cid];
            while(elem){
                fact *= elem; 
                elem--; 
            }
            d_b[rid * n + cid] = fact; 
        }
    }
}

int main(){
    int *a, *d_a, *b, *d_b, n;
    
    printf("Enter n: ");
    scanf("%d", &n);

    int size = n * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);

    printf("Enter the values of the matrix: ");
    for(int i = 0; i < n*n; i++)
        scanf("%d", &a[i]);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(n,n,1);
    kernelOperation <<<1, dimBlock>>> (d_a, d_b, n);

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++)
            printf("%d\t", b[i*n + j]); 
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);

    return 0; 

}