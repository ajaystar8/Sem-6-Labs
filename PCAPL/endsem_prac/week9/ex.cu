#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(int *d_a, int *d_t){
    int k = threadIdx.x; 
    int row = blockIdx.x;
    int width = blockDim.x;
    int transposed_width = gridDim.x; 

    //in d_t[] -> col = row for a transposed matrix
    d_t[k * transposed_width + row] = d_a[row * width + k];
}

int main(){
    int *a, *t, m, n; 
    int *d_a, *d_t; 

    printf("Enter the value of m: ");
    scanf("%d", &m);

    printf("Enter the value of n: ");
    scanf("%d", &n);

    int size = m*n*sizeof(int);

    // 1. Allocating host memory
    a = (int *)malloc(size);
    t = (int *)malloc(size);

    // 2. Taking input for matrix
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);

    // 3. Allocating device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_t, size);

    // 4. Sending memory to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // 5. Kernel Launch
    transpose<<<m,n>>>(d_a, d_t);

    // 6. Sending results from device to host
    hipMemcpy(t, d_t, size, hipMemcpyDeviceToHost);

    // 7. Display result
    printf("Resultant vector is: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++)
            printf("%d\t", t[i*m + j]); 
        printf("\n");
    }

    // 8. Free device memory
    hipFree(d_a);
    hipFree(d_t);

    return 0; 
}