#include <stdio.h>
#include <stdlib.h> 
#include "hip/hip_runtime.h"


__global__ void addRow(int *d_a, int *d_b, int *d_c){
    int k = threadIdx.x; 
    int width = blockDim.x;
    int row = blockIdx.x; 
    d_c[row*width + k] = d_a[row*width + k] + d_b[row*width + k];
}

__global__ void addColumn(int *d_a, int *d_b, int *d_c){
    int k = threadIdx.x; 
    int width = blockDim.x;
    int col = blockIdx.x;
    
    d_c[k*width + col] = d_a[k*width + col] + d_b[k*width + col];
}

__global__ void addElement(int *d_a, int *d_b, int *d_c){
    int elem = threadIdx.x; 
    d_c[elem] = d_a[elem] + d_b[elem];
}

int main(){
    int *a, *b, *c, m, n, size;
    int *d_a, *d_b, *d_c; 

    printf("Enter the value of m: ");
    scanf("%d", &m);

    printf("Enter the value of n: ");
    scanf("%d", &n);

    size = m*n*sizeof(int);

    // allocate host memory

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // take input to fill matrix

    printf("Enter %d elements of Matrix-A: ", m*n);
    for(int i = 0; i < m*n; i++)
        scanf("%d", &a[i]);

    printf("Enter %d elements of Matrix-B: ", m*n);
    for(int i = 0; i < m*n; i++)
        scanf("%d", &b[i]);

    // allocate device memory

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // transfer memory from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch kernel - 1a
    addRow <<<m, n>>> (d_a, d_b, d_c);

    // transfer memory from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // display results

    printf("\nResultant vector by addRow is: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++)
            printf("%d\t", c[i*m + j]); 
        printf("\n");
    }

    // launch kernel - 1b
    addRow <<<n, m>>> (d_a, d_b, d_c);

    // transfer memory from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // display results

    printf("\nResultant vector by addColumn is: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++)
            printf("%d\t", c[i*m + j]); 
        printf("\n");
    }

    // launch kernel - 1c
    addRow <<<1, m*n>>> (d_a, d_b, d_c);

    // transfer memory from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // display results

    printf("\nResultant vector by addElement is: \n");

    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++)
            printf("%d\t", c[i*m + j]); 
        printf("\n");
    }

    // release memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0; 
}