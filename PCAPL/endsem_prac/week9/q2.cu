#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void mulRow(int *d_a, int *d_b, int *d_c, int wa, int wb){
    int rid = threadIdx.x; 
    int sum = 0; 
    for(int cidB = 0; cidB < wb; cidB++){
        sum = 0; 
        for(int k = 0; k < wa; k++)
            sum += d_a[rid*wa + k] * d_b[k*wb + cidB];
        d_c[rid*wb + cidB] = sum; 
    }
}

__global__ void mulColumn(int *d_a, int *d_b, int *d_c, int ha, int wa){
    int cidB = threadIdx.x; 
    int sum = 0; 
    int wb = blockDim.x; 
    for(int ridA = 0; ridA < ha; ridA++){
        sum = 0; 
        for(int k = 0; k < wa; k++)
            sum += d_a[ridA * wa + k] * d_b[k * wb + cidB];
        d_c[ridA*wb + cidB] = sum; 
    }
}

__global__ void mulElement(int *d_a, int *d_b, int *d_c, int wa){
    int ridA = threadIdx.y;
    int cidB = threadIdx.x;
    int wb = blockDim.x; 
    int sum = 0; 
    for(int k = 0; k < wa; k++)
        sum += d_a[ridA * wa + k] * d_b[k * wb + cidB];
    d_c[ridA * wb + cidB] = sum; 
}

int main(){
    int *a, *b, *c, ha, wa, hb, wb; 
    int *d_a, *d_b, *d_c; 

    printf("Enter ha: ");
    scanf("%d", &ha);

    printf("Enter wa: ");
    scanf("%d", &wa);

    printf("Enter hb: ");
    scanf("%d", &hb);

    printf("Enter wb: ");
    scanf("%d", &wb);

    int size1 = ha * wa * sizeof(int);
    int size2 = hb * wb * sizeof(int);
    int ressize = ha * wb * sizeof(int);

    // allocate host memory
    a = (int *)malloc(size1);
    b = (int *)malloc(size2);
    c = (int *)malloc(ressize);

    // take input for matrices
    printf("Enter the elements of Matrix-A: ");
    for(int i = 0; i < ha*wa; i++)
        scanf("%d", &a[i]);
    
    printf("Enter the elements of Matrix-B: ");
    for(int i = 0; i < hb*wb; i++)
        scanf("%d", &b[i]);

    // allocate device memory
    hipMalloc((void **)&d_a, size1);
    hipMalloc((void **)&d_b, size2);
    hipMalloc((void **)&d_c, ressize);

    // send host memory to device
    hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);

    // // launch kernel
    // mulRow <<< 1, ha >>> (d_a, d_b, d_c, wa, wb);

    // // send results to host
    // cudaMemcpy(c, d_c, ressize, cudaMemcpyDeviceToHost);

    // // display results
    // printf("Resultant matrix -mulRow: \n");

    // for(int i = 0; i < ha; i++){
    //     for(int j = 0; j < wb; j++)
    //         printf("%d\t", c[i*ha + j]); 
    //     printf("\n");
    // }

    // // launch kernel
    // mulColumn <<< 1, wb >>> (d_a, d_b, d_c, ha, wa);

    // // send results to host
    // cudaMemcpy(c, d_c, ressize, cudaMemcpyDeviceToHost);

    // // display results
    // printf("Resultant matrix - mulColumn: \n");

    // for(int i = 0; i < ha; i++){
    //     for(int j = 0; j < wb; j++)
    //         printf("%d\t", c[i*ha + j]); 
    //     printf("\n");
    // }

    // launch kernel
    dim3 block_size(wb, ha);
    mulElement <<< 1, block_size >>> (d_a, d_b, d_c, wa);


    // send results to host
    hipMemcpy(c, d_c, ressize, hipMemcpyDeviceToHost);

    // display results
    printf("Resultant matrix - mulElement: \n");

    for(int i = 0; i < ha; i++){
        for(int j = 0; j < wb; j++)
            printf("%d\t", c[i*ha + j]); 
        printf("\n");
    }

    // release memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0; 

}