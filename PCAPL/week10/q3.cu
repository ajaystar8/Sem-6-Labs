#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void border(int *dArr, int m, int n) {
    int row = blockIdx.x / n;
    int col = blockIdx.x % n;
    if(row > 0 && row < m-1 && col > 0 && col < n-1) {
        int n = 1;
        while(pow(2, n) < dArr[blockIdx.x]) n++;
        if(pow(2, n) == dArr[blockIdx.x]) n++;
        int a[10] ,num , i;
        num = pow(2, n) - 1 - dArr[blockIdx.x];
        for(i=0;num>0;i++) {
            a[i]=num%2;
            num=num/2;
        }
        int ans = 0;
        for(i=i-1;i>=0;i--)
            ans=(ans*10)+a[i];
        dArr[blockIdx.x] = ans;
    }
}

int main() {
    int m, n;
    printf("Enter the dimensions of the matrix: ");
    scanf("%d %d", &m, &n);
    int arr[m*n];
    int *dArr;
    printf("Enter the elements of the matrix: ");
    for(int i=0; i<m*n; i++)
        scanf("%d", &arr[i]);
    hipMalloc(&dArr, m*n*sizeof(int));
    hipMemcpy(dArr, arr, m*n*sizeof(int), hipMemcpyHostToDevice);
    border<<<m*n, 1>>>(dArr, m, n);
    hipMemcpy(arr, dArr, m*n*sizeof(int), hipMemcpyDeviceToHost);
    printf("Matrix:\n");
    for(int i=0; i<m*n; i++) {
        if(i%n == 0) printf("\n");
        printf("%d ", arr[i]);
    }
    hipFree(dArr);
    return 0;
}