#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void matrixAddRow(int *d_mat1, int *d_mat2, int *d_ans, int m, int n) {
	int tid=blockIdx.x;
	for(int i=0; i<n; i++)
		d_ans[tid*n+i] = d_mat1[tid*n+i] + d_mat2[tid*n+i];
}

__global__ void matrixAddCol(int *d_mat1, int *d_mat2, int *d_ans, int m, int n) {
	int tid=blockIdx.x;
	for(int i=0; i<n; i++)
		d_ans[tid+n*i] = d_mat1[tid+n*i] + d_mat2[tid+n*i];
}

__global__ void matrixAddEle(int *d_mat1, int *d_mat2, int *d_ans, int m, int n) {
	int tid=blockIdx.x;
	d_ans[tid] = d_mat1[tid] + d_mat2[tid];
}

int main(void) {
	int m, n;

	printf("Enter the row and col size of the matrix: ");
	scanf("%d %d", &m, &n);

	int *mat1, *mat2, *ans;
	int *d_mat1, *d_mat2, *d_ans;

	mat1 = (int*)malloc(m*n*sizeof(int));
	mat2 = (int*)malloc(m*n*sizeof(int));
	ans = (int*)malloc(m*n*sizeof(int));

	hipMalloc((void **)&d_mat1, m*n*sizeof(int));
	hipMalloc((void **)&d_mat2, m*n*sizeof(int));
	hipMalloc((void **)&d_ans, m*n*sizeof(int));

    printf("Enter the elements of the matrix1: ");
		for(int i=0; i<m*n; i++)
			scanf("%d", &mat1[i]);
	printf("Enter the elements of the matrix2: ");
		for(int i=0; i<m*n; i++)
			scanf("%d", &mat2[i]);

	hipMemcpy(d_mat1, mat1, m*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat2, mat2, m*n*sizeof(int), hipMemcpyHostToDevice);

	matrixAddRow<<<m, 1>>>(d_mat1, d_mat2, d_ans, m, n);

	hipMemcpy(ans, d_ans, m*n*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m*n; i++) {
		if(i%n == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}

	matrixAddCol<<<m, 1>>>(d_mat1, d_mat2, d_ans, m, n);

	hipMemcpy(ans, d_ans, m*n*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m*n; i++) {
		if(i%n == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}

	matrixAddEle<<<m*n, 1>>>(d_mat1, d_mat2, d_ans, m, n);

	hipMemcpy(ans, d_ans, m*n*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m*n; i++) {
		if(i%n == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}
	
	hipFree(d_mat1);
	hipFree(d_mat2);
	hipFree(d_ans);

	return 0;
}
