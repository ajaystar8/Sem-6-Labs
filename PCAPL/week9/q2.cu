#include <stdio.h>
#include "hip/hip_runtime.h"

#include <stdlib.h>

/// check start

__global__ void matrixMulRow(int *d_mat1, int *d_mat2, int *d_ans, int m1, int n1, int m2, int n2) {
	int tid=blockIdx.x;
    int sum = 0;
    for(int j=0; j<m1; j++) {
        for(int i=0; i<n1; i++) {
            sum += d_mat1[tid*n1+i] * d_mat2[j+n2*i];
        }
        d_ans[tid*m1+j] = sum;
        sum = 0;
    }
}

// work on second row  2 3 3 2 1 2 3 4 5 6
__global__ void matrixMulCol(int *d_mat1, int *d_mat2, int *d_ans, int m1, int n1, int m2, int n2) {
	int tid=blockIdx.x;
    int sum = 0;
    for(int j=0; j<m1; j++) {
        for(int i=0; i<m2; i++) {
            sum += d_mat1[j*m2+i] * d_mat2[tid+n2*i];
            if(tid == 0) {
                printf("%d - %d\n", d_mat1[j*m2+i], d_mat2[tid+n2*i]);
            }
        }
        d_ans[tid+n1*j] = sum;
        sum = 0;
    }
}

// __global__ void matrixMulEle(int *d_mat1, int *d_mat2, int *d_ans, int m1, int n1, int m2, int n2) {
// 	int tid=blockIdx.x;
//     int row = tid/n1, col = tid%n2;
//     int sum = 0;
// 	for(int i=0; i<n1; i++)
//         sum += d_mat1[n1*row+i] * d_mat2[n1*i+col];
//     d_ans[tid] = sum;
// }

/// check end

int main(void) {
	int m1, n1, m2, n2;

	printf("Enter the row and col size of the matrix: ");
	scanf("%d %d %d %d", &m1, &n1, &m2, &n2);

    if(n1 != m2) {
        printf("Matrices are not compatible for multiplication.\n");
        exit(0);
    }


	int *mat1, *mat2, *ans;
	int *d_mat1, *d_mat2, *d_ans;

	mat1 = (int*)malloc(m1*n1*sizeof(int));
	mat2 = (int*)malloc(m2*n2*sizeof(int));
	ans = (int*)malloc(m1*n2*sizeof(int));

	hipMalloc((void **)&d_mat1, m1*n1*sizeof(int));
	hipMalloc((void **)&d_mat2, m2*n2*sizeof(int));
	hipMalloc((void **)&d_ans, m1*n2*sizeof(int));

    printf("Enter the elements of the matrix1: ");
		for(int i=0; i<m1*n1; i++)
			scanf("%d", &mat1[i]);
	printf("Enter the elements of the matrix2: ");
		for(int i=0; i<m2*n2; i++)
			scanf("%d", &mat2[i]);

	hipMemcpy(d_mat1, mat1, m1*n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat2, mat2, m2*n2*sizeof(int), hipMemcpyHostToDevice);

    /// check start

	// matrixMulRow<<<m1, 1>>>(d_mat1, d_mat2, d_ans, m1, n1, m2, n2);

	// cudaMemcpy(ans, d_ans, m1*n2*sizeof(int), cudaMemcpyDeviceToHost);
	
	// printf("Final matrix is:\n");
	// for(int i=0; i<m1*n2; i++) {
	// 	if(i%n2 == 0)
	// 		printf("\n");
	// 	printf("%d ", ans[i]);
	// }

	matrixMulCol<<<m1, 1>>>(d_mat1, d_mat2, d_ans, m1, n1, m2, n2);

	hipMemcpy(ans, d_ans, m1*n2*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Final matrix is:\n");
	for(int i=0; i<m1*n2; i++) {
		if(i%n2 == 0)
			printf("\n");
		printf("%d ", ans[i]);
	}

	// matrixMulEle<<<m1*n2, 1>>>(d_mat1, d_mat2, d_ans, m1, n1, m2, n2);

	// cudaMemcpy(ans, d_ans, m1*n2*sizeof(int), cudaMemcpyDeviceToHost);
	
	// printf("Final matrix is:\n");
	// for(int i=0; i<m1*n2; i++) {
	// 	if(i%n2 == 0)
	// 		printf("\n");
	// 	printf("%d ", ans[i]);
	// }
	
	hipFree(d_mat1);
	hipFree(d_mat2);
	hipFree(d_ans);

    /// check end

	return 0;
}